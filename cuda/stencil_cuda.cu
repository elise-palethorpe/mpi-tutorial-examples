#include "hip/hip_runtime.h"
/*
 * 2D stencil code
 */

#include <hip/hip_runtime.h>
#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>

/* Input parameters */
int n;
int niters;

/* row-major order */
#define ind(i,j) ((j)*(bx+2)+(i))

__global__
void init_grid(double *anew, double *aold)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    anew[i] = 0.0;
    aold[i] = 0.0;
}

__global__
void update_source(double *aold, int bx, int by, int i, int j, double energy)
{
    aold[ind(i,j)] += energy;
}

__global__
void update_grid(double *anew, double *aold, int bx, int by)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    if (i <= bx && j <= by) {
        anew[ind(i, j)] = aold[ind(i, j)] / 2.0 +
                          (aold[ind(i - 1, j)] + aold[ind(i + 1, j)] +
                           aold[ind(i, j - 1)] + aold[ind(i, j + 1)]) / 4.0 / 2.0;
    }
}

__global__
void gather_heat(double *aold, int bx, int by, double *heat)
{
    *heat = 0.0;
    for (int i = 1; i < bx + 1; i++)
        for (int j = 1; j < by + 1; j++)
            *heat += aold[ind(i, j)];
}

int main(int argc, char **argv)
{
    MPI_Init(&argc, &argv);

    /* get input parameters from command line options */
    if (argc != 3) {
        printf("usage: %s <n> <niters>\n", argv[0]);
        return 0;
    }

    n = atoi(argv[1]);          /* n x n grid */
    niters = atoi(argv[2]);     /* number of iterations */

    /* domain size */
    int bx, by;
    bx = n;
    by = n;

    /* NOTE: use 8x8 block size. */
    assert(bx % 8 == 0);
    assert(by % 8 == 0);

    /* energy to be injected per iteration per source */
    int energy = 1.0;

    /* initialize three heat sources */
#define NSOURCES 3
    int sources[NSOURCES][2];
    sources[0][0] = n / 2;
    sources[0][1] = n / 2;
    sources[1][0] = n / 3;
    sources[1][1] = n / 3;
    sources[2][0] = n * 4 / 5;
    sources[2][1] = n * 8 / 9;

    /* allocate working arrays & communication buffers.
     * NOTE: Include 1-wide hallo zones on each side. */
    double *aold, *anew;
    hipMalloc(&anew, (bx + 2) * (by + 2) * sizeof(double));
    hipMalloc(&aold, (bx + 2) * (by + 2) * sizeof(double));
    /* initialize */
    init_grid<<<by + 2, bx + 2>>>(anew, aold);

    /* prepare kernel launching dimesnions */
    dim3 block_dim = dim3(8, 8);
    dim3 grid_dim = dim3(bx/8, by/8);
    if (bx % 8) grid_dim.x++;
    if (by % 8) grid_dim.y++;

    double t_begin = MPI_Wtime();
    for (int iter = 0; iter < niters; ++iter) {
        /* refresh heat sources */
        for (int i = 0; i < NSOURCES; ++i) {
            update_source<<<1, 1>>>(aold, bx, by, sources[i][0], sources[i][1], energy);
        }

        /* update grid points */
        update_grid<<<grid_dim, block_dim>>>(anew, aold, bx, by);

        /* swap working arrays */
        double *tmp = anew;
        anew = aold;
        aold = tmp;
    }

    double last_heat;
    double *heat;
    hipMalloc(&heat, sizeof(double));
    gather_heat<<<1, 1>>>(aold, bx, by, heat);
    hipMemcpy(&last_heat, heat, sizeof(double), hipMemcpyDeviceToHost);
    hipFree(heat);

    double t_end = MPI_Wtime();

    /* free working arrays and communication buffers */
    hipFree(aold);
    hipFree(anew);

    /* get final heat in the system */
    printf("last heat: %f\n", last_heat);
    printf("    Total computation time: %.6f sec.\n", t_end - t_begin);

    MPI_Finalize();
    return 0;
}
